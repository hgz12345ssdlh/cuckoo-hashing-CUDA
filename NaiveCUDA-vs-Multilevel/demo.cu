#include "hip/hip_runtime.h"
#include "cuckoo-cuda-naive.cuh"
#include "cuckoo-cuda-multi.cuh"
#include <map>
#include <cstdlib>
#include <cmath>
#include <cstdint>


/** Random input generator. */
// static void
// gen_rnd_input(uint32_t * const vals, const int n, const uint32_t limit) {
//     std::map<uint32_t, bool> val_map;
//     int count = 0;
//     while (count < n) {
//         uint32_t val = (rand() % (limit - 1)) + 1;
//         if (val_map.find(val) != val_map.end())
//             continue;
//         val_map[val] = true;
//         vals[count] = val;
//         count++;
//     }
// }


/**
 *
 * Main entrance for the demo.
 * 
 * Prerequirests: we assume
 *   1. Value range do not cover EMPTY_CELL (i.e. 0).
 *   2. Value range do not exceed value-field width.
 *   3. No repeated values inserted (so we skipped duplication check).
 *   4. Size must be a multiple of BUCKET_SIZE.
 *
 * Currently supported types:
 *   uint[8, 16, 32]_t
 *   
 */
int
main(void) {

    // DEMO of CUDA naive implementation.
    std::cout << "CUDA naive implementation DEMO -->" << std::endl << std::endl;
    {
        CuckooHashTableCuda_Naive<uint32_t> table_cuda(8, 4 * ceil(log2((double) 8)), 3);
        table_cuda.show_content();

        std::cout << "Insert 6 values -" << std::endl;
        // uint32_t vals_to_insert[8];
        // gen_rnd_input(vals_to_insert, 8, 0x1 << 30);
        uint32_t vals_to_insert[6] = {5, 6, 7, 8, 9, 10};
        table_cuda.insert_vals(vals_to_insert, 6, 0);
        table_cuda.show_content();

        std::cout << "Delete values [0..3] -" << std::endl;
        uint32_t vals_to_delete[3];
        for (int i = 0; i < 3; ++i)
            vals_to_delete[i] = vals_to_insert[i];
        table_cuda.delete_vals(vals_to_delete, 3);
        table_cuda.show_content();

        std::cout << "Lookup values [2..4] -" << std::endl;
        uint32_t vals_to_lookup[2];
        for (int i = 0; i < 2; ++i)
            vals_to_lookup[i] = vals_to_insert[i + 2];
        bool results[2];
        table_cuda.lookup_vals(vals_to_lookup, results, 2);
        std::cout << "Results - ";
        for (int i = 0; i < 2; ++i)
            std::cout << results[i] << " ";
        std::cout << std::endl;
        table_cuda.show_content();
    }

    // DEMO of CUDA multi-level implementation.
    std::cout << "CUDA multi-level implementation DEMO -->" << std::endl << std::endl;
    {
        CuckooHashTableCuda_Multi<uint32_t> table_cuda(8, 4 * ceil(log2((double) 8)), 3);
        table_cuda.show_content();

        std::cout << "Insert 6 values -" << std::endl;
        // uint32_t vals_to_insert[8];
        // gen_rnd_input(vals_to_insert, 8, 0x1 << 30);
        uint32_t vals_to_insert[6] = {5, 6, 7, 8, 9, 10};
        table_cuda.insert_vals(vals_to_insert, 6);
        table_cuda.show_content();

        std::cout << "Delete values [0..3] -" << std::endl;
        uint32_t vals_to_delete[3];
        for (int i = 0; i < 3; ++i)
            vals_to_delete[i] = vals_to_insert[i];
        table_cuda.delete_vals(vals_to_delete, 3);
        table_cuda.show_content();

        std::cout << "Lookup values [2..4] -" << std::endl;
        uint32_t vals_to_lookup[2];
        for (int i = 0; i < 2; ++i)
            vals_to_lookup[i] = vals_to_insert[i + 2];
        bool results[2];
        table_cuda.lookup_vals(vals_to_lookup, results, 2);
        std::cout << "Results - ";
        for (int i = 0; i < 2; ++i)
            std::cout << results[i] << " ";
        std::cout << std::endl;
        table_cuda.show_content(); 
    }

    return 0;
}
